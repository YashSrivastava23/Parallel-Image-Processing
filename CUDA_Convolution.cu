#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>

#define WIDTH 1024
#define HEIGHT 1024
#define KERNEL_SIZE 3

__global__ void convolution(const float* image, const float* kernel, float* result, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float value = 0.0f;
    for (int ky = -KERNEL_SIZE / 2; ky <= KERNEL_SIZE / 2; ++ky) {
        for (int kx = -KERNEL_SIZE / 2; kx <= KERNEL_SIZE / 2; ++kx) {
            int ix = min(max(x + kx, 0), width - 1);
            int iy = min(max(y + ky, 0), height - 1);
            value += image[iy * width + ix] * kernel[(ky + KERNEL_SIZE / 2) * KERNEL_SIZE + (kx + KERNEL_SIZE / 2)];
        }
    }
    result[y * width + x] = value;
}

void checkCudaError(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        std::cerr << msg << " Error: " << hipGetErrorString(error) << std::endl;
        throw std::runtime_error(msg);
    }
}

void cudaConvolution(const float* h_image, const float* h_kernel, float* h_result, int width, int height) {
    float *d_image, *d_kernel, *d_result;

    size_t imageSize = width * height * sizeof(float);
    size_t kernelSize = KERNEL_SIZE * KERNEL_SIZE * sizeof(float);

    checkCudaError(hipMalloc(&d_image, imageSize), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_kernel, kernelSize), "Failed to allocate device memory for kernel");
    checkCudaError(hipMalloc(&d_result, imageSize), "Failed to allocate device memory for result");

    checkCudaError(hipMemcpy(d_image, h_image, imageSize, hipMemcpyHostToDevice), "Failed to copy image to device");
    checkCudaError(hipMemcpy(d_kernel, h_kernel, kernelSize, hipMemcpyHostToDevice), "Failed to copy kernel to device");

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    convolution<<<gridSize, blockSize>>>(d_image, d_kernel, d_result, width, height);
    checkCudaError(hipGetLastError(), "Kernel launch failed");

    checkCudaError(hipMemcpy(h_result, d_result, imageSize, hipMemcpyDeviceToHost), "Failed to copy result to host");

    checkCudaError(hipFree(d_image), "Failed to free device memory for image");
    checkCudaError(hipFree(d_kernel), "Failed to free device memory for kernel");
    checkCudaError(hipFree(d_result), "Failed to free device memory for result");
}

void logCudaDeviceProperties() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA Devices: " << deviceCount << std::endl;
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << "Device " << i << ": " << props.name << std::endl;
        std::cout << "  Compute capability: " << props.major << "." << props.minor << std::endl;
        std::cout << "  Total memory: " << props.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    }
}

void performCudaImageTransformation(const float* h_image, float* h_result, int width, int height) {
    float* d_image, *d_result;
    size_t imageSize = width * height * sizeof(float);

    checkCudaError(hipMalloc(&d_image, imageSize), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_result, imageSize), "Failed to allocate device memory for result");

    checkCudaError(hipMemcpy(d_image, h_image, imageSize, hipMemcpyHostToDevice), "Failed to copy image to device");

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    // Dummy transformation kernel launch
    convolution<<<gridSize, blockSize>>>(d_image, d_image, d_result, width, height);
    checkCudaError(hipGetLastError(), "Kernel launch failed");

    checkCudaError(hipMemcpy(h_result, d_result, imageSize, hipMemcpyDeviceToHost), "Failed to copy result to host");

    checkCudaError(hipFree(d_image), "Failed to free device memory for image");
    checkCudaError(hipFree(d_result), "Failed to free device memory for result");
}

void performCudaEdgeDetection(const float* h_image, float* h_result, int width, int height) {
    float* d_image, *d_result;
    size_t imageSize = width * height * sizeof(float);

    checkCudaError(hipMalloc(&d_image, imageSize), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_result, imageSize), "Failed to allocate device memory for result");

    checkCudaError(hipMemcpy(d_image, h_image, imageSize, hipMemcpyHostToDevice), "Failed to copy image to device");

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    // Dummy edge detection kernel launch
    convolution<<<gridSize, blockSize>>>(d_image, d_image, d_result, width, height);
    checkCudaError(hipGetLastError(), "Kernel launch failed");

    checkCudaError(hipMemcpy(h_result, d_result, imageSize, hipMemcpyDeviceToHost), "Failed to copy result to host");

    checkCudaError(hipFree(d_image), "Failed to free device memory for image");
    checkCudaError(hipFree(d_result), "Failed to free device memory for result");
}